#include "hip/hip_runtime.h"
#include "operators/evtl.hpp"

extern "C" {
#include <check.h>
#include <math.h>
#include <stdio.h>
#include <string.h>

#include "config.h"
#include "sigpt.h"
#include "util.h"
}

#define NITEMS (42)

/* TODO why do we have such big differences ??? at least with 0.05 it' ok, but... */
#define FLOAT_EQUALS(x, y) (fabs((x) - (y)) < 0.05)

#define EVTL_TEST(name, fin, fexpected) \
START_TEST(name) \
{ \
    sigpt_t *in, *out; \
 \
    int in_n = read_signal_file(SIG_PATH "/" fin, &in); \
    fail_unless(in_n != -1); \
 \
    thrust::device_vector<sigpt_t> vin(in, in + in_n); \
    thrust::device_vector<sigpt_t> vout(2 * in_n); \
 \
    stl_evtl(vin, vout); \
 \
    thrust::host_vector<sigpt_t> host_out(vout); \
 \
    int out_n = read_signal_file(SIG_PATH "/" fexpected, &out); \
    fail_unless(out_n != -1); \
 \
    fail_unless(sigcmp(out, host_out.data(), MAX(out_n, host_out.size())) == 0); \
 \
    free(in); \
    free(out); \
} \
END_TEST

static int
sigcmp(const sigpt_t *lhs, const sigpt_t *rhs, int n)
{
    int i = 0, j = 0;
    const sigpt_t *l, *r;

    /* TODO: One signal shorter than other. */

    while (i < n && j < n) {
        l = lhs + i;
        r = rhs + j;

        /* Note: dy is ignored for now because it's missing in the breach signal traces. */
        if (!FLOAT_EQUALS(l->t, r->t) ||
                !FLOAT_EQUALS(l->y, r->y)) {
            fprintf(stderr, "lhs[%d]: { t: %f, y: %f, dy: %f } != "
                            "rhs[%d]: { t: %f, y: %f, dy: %f }\n",
                    i, l->t, l->y, l->dy,
                    j, r->t, r->y, r->dy);
            return -1;
        }

        do {
            i++;
        } while (i < n && FLOAT_EQUALS(l->y, lhs[i].y));

        do {
            j++;
        } while (j < n && FLOAT_EQUALS(r->y, rhs[j].y));

    }

    return 0;
}

START_TEST(test_sanity)
{
    sigpt_t *a = sigpt_random(42, NITEMS);
    sigpt_t *c = (sigpt_t *)calloc(2 * NITEMS,sizeof(sigpt_t));

    thrust::device_vector<sigpt_t> in(a, a + NITEMS);
    thrust::device_vector<sigpt_t> out(c, c + 2 * NITEMS);

    stl_evtl(in, out);

    free(a);
    free(c);
}
END_TEST

EVTL_TEST(test_sig1, "sig05.trace", "ev_sig05.breach.trace")
EVTL_TEST(test_sig2, "sig06.trace", "ev_sig06.breach.trace")

static Suite *
create_suite(void)
{
    Suite *s = suite_create(__FILE__);
    TCase *tc_core = tcase_create("core");

    tcase_add_test(tc_core, test_sanity);
    tcase_add_test(tc_core, test_sig1);
    tcase_add_test(tc_core, test_sig2);

    suite_add_tcase(s, tc_core);

    return s;
}

int
main(int argc __attribute__ ((unused)),
     char **argv __attribute__ ((unused)))
{
    int number_failed;
    Suite *s = create_suite();
    SRunner *sr = srunner_create(s);
    srunner_run_all(sr, CK_NORMAL);
    number_failed = srunner_ntests_failed(sr);
    srunner_free(sr);

    return (number_failed == 0) ? EXIT_SUCCESS : EXIT_FAILURE;
}
