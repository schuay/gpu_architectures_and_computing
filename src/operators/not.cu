#include "hip/hip_runtime.h"
#include "not.hpp"

void
stl_not(const thrust::device_vector<sigpt_t> &in,
        thrust::device_vector<sigpt_t> &out)
{
}

/**
 * sizeof(out) == sizeof(in).
 */
__global__ void
stl_not(const sigpt_t *in, sigpt_t *out, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        sigpt_t s = in[i];
        s.y *= -1.f;
        s.dy *= -1.f;
        out[i] = s;
    }
}
