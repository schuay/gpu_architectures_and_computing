#include "hip/hip_runtime.h"
#include "not.hpp"

#include "globals.h"

/**
 * sizeof(out) == sizeof(in).
 */
__global__ void
stl_not(const sigpt_t *in, sigpt_t *out, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        sigpt_t s = in[i];
        s.y *= -1.f;
        s.dy *= -1.f;
        out[i] = s;
    }
}

void
stl_not(const thrust::device_ptr<sigpt_t> &in,
        thrust::device_ptr<sigpt_t> *out, int n)
{
    const sigpt_t *ptr_in = thrust::raw_pointer_cast(in.get());
    thrust::device_ptr<sigpt_t> ptr_out = thrust::device_malloc<sigpt_t>(n);

    stl_not<<<NBLOCKS, NTHREADS>>>(ptr_in, thrust::raw_pointer_cast(ptr_out), n);

    *out = ptr_out;
}
