#include "hip/hip_runtime.h"
#include "evtl.hpp"

#include <thrust/scan.h>

#include "globals.h"

#define CUDA_MAX(a, b) (((a) > (b)) * (a) + ((a) <= (b)) * (b))

struct sigpt_max : public thrust::binary_function<sigpt_t, sigpt_t, sigpt_t>
{
    __device__ sigpt_t
    operator()(const sigpt_t &lhs, const sigpt_t &rhs) const
    {
        return (sigpt_t) {rhs.t, CUDA_MAX(lhs.y, rhs.y), 0};
    }
};

/* Stores all elements zs[i] at zs_intersect[i * 2] and uses zs and ys to
 * interpolate points at zs_intersect[i * 2 + 1] if needed.
 * For every position in zs_intersect that contains a valid point the
 * according position in cs is set to 1.
 * ys: the original signal.
 * zs: the signal after the initial scan.
 */
__global__ static void
evtl_intersect(const sigpt_t *ys,
               sigpt_t *zs,
               sigpt_t *zs_intersect,
               char *cs,
               const int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        cs[i * 2] = 1;
        zs_intersect[i * 2] = zs[i];
        // FIXME: Branches are bad.
        if (i < n - 1 && zs[i].y > zs[i + 1].y) {
            cs[i * 2 + 1] = 1;
            zs_intersect[i * 2 + 1].t = zs[i + 1].t +
                (zs[i + 1].t - zs[i].t) *
                (zs[i + 1].y - ys[i + 1].y) / (ys[i + 1].y - ys[i].y);
            zs_intersect[i * 2 + 1].y = zs[i + 1].y;
        }
    }
}

/* Store each element zs[i] where cs[i] is 1 in zs_final[fs[i]]. */
__global__ static void
evtl_compact(const sigpt_t *zs,
             sigpt_t *zs_final,
             const char *cs,
             const size_t *fs,
             const int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // FIXME: Branches are bad.
        if (cs[i] == 1) {
            zs_final[fs[i]] = zs[i];
        }
    }
}

void
stl_evtl(const thrust::device_ptr<sigpt_t> &in,
         const int nin,
         thrust::device_ptr<sigpt_t> *out,
         int *nout)
{
    const int dnout = 2 * nin;
    thrust::device_ptr<sigpt_t> dout = thrust::device_malloc<sigpt_t>(dnout);

    thrust::reverse_iterator<thrust::device_ptr<sigpt_t> > rin(in + nin);
    thrust::reverse_iterator<thrust::device_ptr<sigpt_t> > rout(dout + dnout);

    /* Propagate max rubustness backwards */
    thrust::inclusive_scan(rin, rin + nin, rout + nin, sigpt_max()); 

    const sigpt_t *ys = in.get();

    thrust::device_vector<sigpt_t> out_intersect(dnout);
    sigpt_t *zs = thrust::raw_pointer_cast(out_intersect.data());

    sigpt_t *zs_final = dout.get();

    /* Used positions in zs (out_intersect) are marked here.
     * This amounts to all positions where there was an original value plus
     * those where we had to interpolate a value. */
    thrust::device_vector<char> used(dnout, 0);
    char *cs = thrust::raw_pointer_cast(used.data());
    evtl_intersect<<<NBLOCKS, NTHREADS>>>(ys, zs_final, zs, cs, nin);

    thrust::device_vector<size_t> positions(dnout, 0);
    /* Scan over the used positions in cs and store the result in positions
     * For a point in zs[i], positions[i] will now contain the index at
     * which that point should end up in the result array. */
    thrust::exclusive_scan(used.cbegin(), used.cend(), positions.begin(), 0, thrust::plus<size_t>()); 

    size_t *fs = thrust::raw_pointer_cast(positions.data());
    evtl_compact<<<NBLOCKS, NTHREADS>>>(zs, zs_final, cs, fs, dnout);

    *out = dout;
    *nout = positions.back(); /* Note that we don't actually resize here. */
}
