#include "hip/hip_runtime.h"
#include "evtl.hpp"

#include <thrust/scan.h>

#include "globals.h"

#define CUDA_MAX(a, b) (((a) > (b)) * (a) + ((a) <= (b)) * (b))

struct sigpt_max : public thrust::binary_function<sigpt_t, sigpt_t, sigpt_t>
{
    __device__ sigpt_t
    operator()(const sigpt_t &lhs, const sigpt_t &rhs) const
    {
        return (sigpt_t) {rhs.t, CUDA_MAX(lhs.y, rhs.y), 0};
    }
};

__global__ static void
evtl_intersect(const sigpt_t *ys,
               sigpt_t *zs,
               sigpt_t *zs_intersect,
               char *cs,
               const int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        cs[i * 2] = 1;
        zs_intersect[i * 2] = zs[i];
        // FIXME: Branches are bad.
        if (i < n - 1 && zs[i].y > zs[i + 1].y) {
            cs[i * 2 + 1] = 1;
            zs_intersect[i * 2 + 1].t = zs[i + 1].t +
                (zs[i + 1].t - zs[i].t) *
                (zs[i + 1].y - ys[i + 1].y) / (ys[i + 1].y - ys[i].y);
            zs_intersect[i * 2 + 1].y = zs[i + 1].y;
        }
    }
}

__global__ static void
evtl_compact(const sigpt_t *zs,
             sigpt_t *zs_final,
             const char *cs,
             const size_t *fs,
             const int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // FIXME: Branches are bad.
        if (cs[i] == 1) {
            zs_final[fs[i]] = zs[i];
        }
    }
}

void
stl_evtl(const thrust::device_ptr<sigpt_t> &in,
         const int nin,
         thrust::device_ptr<sigpt_t> *out,
         int *nout)
{
    const int dnout = 2 * nin;
    thrust::device_ptr<sigpt_t> dout = thrust::device_malloc<sigpt_t>(dnout);

    thrust::reverse_iterator<thrust::device_ptr<sigpt_t> > rin(in + nin);
    thrust::reverse_iterator<thrust::device_ptr<sigpt_t> > rout(dout + dnout);

    thrust::inclusive_scan(rin, rin + nin, rout + nin, sigpt_max()); 

    const sigpt_t *ys = in.get();

    thrust::device_vector<sigpt_t> out_intersect(dnout);
    sigpt_t *zs = thrust::raw_pointer_cast(out_intersect.data());

    sigpt_t *zs_final = dout.get();

    thrust::device_vector<char> used(dnout, 0);
    char *cs = thrust::raw_pointer_cast(used.data());
    evtl_intersect<<<NBLOCKS, NTHREADS>>>(ys, zs_final, zs, cs, nin);

    thrust::device_vector<size_t> positions(dnout, 0);
    thrust::exclusive_scan(used.cbegin(), used.cend(), positions.begin(), 0, thrust::plus<size_t>()); 

    size_t *fs = thrust::raw_pointer_cast(positions.data());
    evtl_compact<<<NBLOCKS, NTHREADS>>>(zs, zs_final, cs, fs, dnout);

    *out = dout;
    *nout = positions.back(); /* Note that we don't actually resize here. */
}
