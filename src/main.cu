#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>

extern "C" {
#include "util.h"
}

#define NBLOCKS (256)
#define NTHREADS (256)

#define NITEMS (256 * 257)

#define checkCudaError(val) do { _checkCudaError((val), #val, __FILE__, __LINE__); } while (0)

bool
_checkCudaError(hipError_t result, const char *func, const char *file, int line)
{
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        return true;
    } else {
        return false;
    }
}

__global__ void simpleAnd(const float *lhs, const float *rhs, float *result, int nitems)
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = tid; i < nitems; i += blockDim.x * gridDim.x) {
		result[i] = ((lhs[i] + rhs[i]) >= 1.f) ? 1.f : 0.f;
	}
}

int
main(int argc, char **argv)
{
    printf("Hello World...\n");

    float *a = random_array(42, NITEMS);
    float *b = random_array(43, NITEMS);
    float *c = (float *)calloc(NITEMS, sizeof(float));
    float *devA, *devB, *devC;

    checkCudaError(hipMalloc((void **)&devA, NITEMS * sizeof(float)));
    checkCudaError(hipMalloc((void **)&devB, NITEMS * sizeof(float)));
    checkCudaError(hipMalloc((void **)&devC, NITEMS * sizeof(float)));

    checkCudaError(hipMemcpy(devA, a, NITEMS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(devB, b, NITEMS * sizeof(float), hipMemcpyHostToDevice));

    printf("Launching kernel...\n");
    simpleAnd<<<NBLOCKS, NTHREADS>>>(devA, devB, devC, NITEMS);

    checkCudaError(hipMemcpy(a, devA, NITEMS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(b, devB, NITEMS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(c, devC, NITEMS * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < 20; i++) {
    	printf("%f & %f = %f\n", a[i], b[i], c[i]);
    }

    checkCudaError(hipFree(devA));
    checkCudaError(hipFree(devB));
    checkCudaError(hipFree(devC));

    free(a);
    free(b);
    free(c);

    return 0;
}
