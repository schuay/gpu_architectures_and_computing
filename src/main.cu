#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

extern "C" {
#include "sigpt.h"
#include "util.h"
}

#include "operators/and.hpp"
#include "operators/evtl.hpp"
#include "operators/or.hpp"
#include "globals.h"

#define NITEMS (256 * 257)
#define TESTFILES_PATH "matlab/traces/"

#define checkCudaError(val) do { _checkCudaError((val), #val, __FILE__, __LINE__); } while (0)


/* TODO: Handle multiple GPUs. */

static bool
_checkCudaError(hipError_t result,
                const char *func,
                const char *file,
                int line)
{
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        return true;
    } else {
        return false;
    }
}

static void
or_test(const char* sig1_filename,
        const char* sig2_filename,
        const char* result_filename)
{
    sigpt_t *a;
    sigpt_t *b;

    int a_n = read_signal_file(sig1_filename, &a);
    int b_n = read_signal_file(sig2_filename, &b);

    if (a_n == 0 || b_n == 0) {
        fprintf(stderr, "couldn't open one of the test files\n");
        return;
    }

    thrust::device_vector<sigpt_t> sig1(a, a + a_n);
    thrust::device_vector<sigpt_t> sig2(b, b + b_n);

    thrust::device_ptr<sigpt_t> d_result;
    int nout;

    hipEvent_t start, stop;
    float elapsedTime;


    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));
    checkCudaError(hipEventRecord(start, 0));

    stl_or(&sig1[0], sig1.size(), &sig2[0], sig2.size(), &d_result, &nout);

    checkCudaError(hipEventRecord(stop, 0));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    printf("\tElapsed time: %f ms\n", elapsedTime);

    thrust::host_vector<sigpt_t> result(d_result, d_result + nout);

    write_signal_file(result_filename,
            result.data(), result.size());

    thrust::device_free(d_result);

    free(a);
    free(b);
}

static void
and_test(const char* sig1_filename,
         const char* sig2_filename,
         const char* result_filename)
{
    sigpt_t *a;
    sigpt_t *b;

    int a_n = read_signal_file(sig1_filename, &a);
    int b_n = read_signal_file(sig2_filename, &b);

    if (a_n == 0 || b_n == 0) {
        fprintf(stderr, "couldn't open one of the test files\n");
        return;
    }

    thrust::device_vector<sigpt_t> sig1(a, a + a_n);
    thrust::device_vector<sigpt_t> sig2(b, b + b_n);

    thrust::device_ptr<sigpt_t> d_result;
    int nout;

    hipEvent_t start, stop;
    float elapsedTime;


    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));
    checkCudaError(hipEventRecord(start, 0));

    stl_and(&sig1[0], sig1.size(), &sig2[0], sig2.size(), &d_result, &nout);

    checkCudaError(hipEventRecord(stop, 0));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    printf("\tElapsed time: %f ms\n", elapsedTime);

    thrust::host_vector<sigpt_t> result(d_result, d_result + nout);

    write_signal_file(result_filename,
            result.data(), result.size());

    thrust::device_free(d_result);

    free(a);
    free(b);
}

static void
eventually_test(const char* sig_filename,
                const char* result_filename)
{
    sigpt_t *a;
    sigpt_t *b;

    int a_n = read_signal_file(sig_filename, &a);
    if (a_n > 0) {
        thrust::device_vector<sigpt_t> in(a, a + a_n);
        thrust::device_ptr<sigpt_t> out;
        int nout;

        hipEvent_t start, stop;
        float elapsedTime;


        checkCudaError(hipEventCreate(&start));
        checkCudaError(hipEventCreate(&stop));
        checkCudaError(hipEventRecord(start, 0));

        stl_evtl(&in[0], in.size(), &out, &nout);

        checkCudaError(hipEventRecord(stop, 0));
        checkCudaError(hipEventSynchronize(stop));
        checkCudaError(hipEventElapsedTime(&elapsedTime, start, stop));
        checkCudaError(hipEventDestroy(start));
        checkCudaError(hipEventDestroy(stop));

        printf("\tElapsed time: %f ms\n", elapsedTime);


        b = (sigpt_t *)calloc(a_n * 2, sizeof(sigpt_t));
        if (b == NULL)
            return;

        for (int i = 0; i < nout; i++)
            b[i] = out[i];

        write_signal_file(result_filename, b, nout);

        thrust::device_free(out);

        free(a);
        free(b);
    } else {
        fprintf(stderr, "couldn't open test file\n");
    }
}

int
main(int argc, char **argv)
{
    and_test(TESTFILES_PATH "sig01.trace",
             TESTFILES_PATH "sig02.trace",
             TESTFILES_PATH "and_sig01_sig02.gpu.trace");
    and_test(TESTFILES_PATH "sig03.trace",
             TESTFILES_PATH "sig04.trace",
             TESTFILES_PATH "and_sig03_sig04.gpu.trace");

    or_test(TESTFILES_PATH "sig01.trace",
            TESTFILES_PATH "sig02.trace",
            TESTFILES_PATH "or_sig01_sig02.gpu.trace");
    or_test(TESTFILES_PATH "sig03.trace",
            TESTFILES_PATH "sig04.trace",
            TESTFILES_PATH "or_sig03_sig04.gpu.trace");

    eventually_test(TESTFILES_PATH "sig05.trace",
                    TESTFILES_PATH "ev_sig05.gpu.trace");
    eventually_test(TESTFILES_PATH "sig06.trace",
                    TESTFILES_PATH "ev_sig06.gpu.trace");

    return 0;
}
