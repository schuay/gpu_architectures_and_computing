#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/merge.h>
#include <thrust/scan.h>
#include <thrust/unique.h>

extern "C" {
#include "sigpt.h"
}

#define NBLOCKS (256)
#define NTHREADS (256)

#define NITEMS (256 * 257)

#define FLOAT_DELTA (0.000000001f)

#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define CUDA_MAX(a, b) (((a) > (b)) * (a) + ((a) <= (b)) * (b))

#define checkCudaError(val) do { _checkCudaError((val), #val, __FILE__, __LINE__); } while (0)

/* TODO: Handle multiple GPUs. */

bool
_checkCudaError(hipError_t result, const char *func, const char *file, int line)
{
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        return true;
    } else {
        return false;
    }
}

/**
 * sizeof(out) == sizeof(in).
 */
__global__ void
stl_not(const sigpt_t *in, sigpt_t *out, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        sigpt_t s = in[i];
        s.y *= -1.f;
        s.dy *= -1.f;
        out[i] = s;
    }
}

#define FLAG_LHS (1 << 0)
#define FLAG_RHS (1 << 1)
#define FLAG_ISC (1 << 2)

typedef struct {
    float t;        /**< The time value. */
    int i;          /**< The original index. */
    int assoc_i;    /**< The associated index of the other signal. */
    int flags;
} seqpt_t;

struct seqpt_less : public thrust::binary_function<seqpt_t, seqpt_t, bool>
{
    __device__ bool
    operator()(const seqpt_t &lhs, const seqpt_t &rhs) const
    {
        return lhs.t < rhs.t;
    }
};

struct sigpt_min : public thrust::binary_function<sigpt_t, sigpt_t, sigpt_t>
{
    __device__ sigpt_t
    operator()(const sigpt_t &lhs, const sigpt_t &rhs) const
    {
        sigpt_t r = rhs;
        sigpt_t s = lhs;

        const int is_rhs_min = (r.y <= s.y) != 0;

        s.y += is_rhs_min * (r.y - s.y);
        s.dy += is_rhs_min * (r.dy - s.dy);

        return s;
    }
};

/**
 * Given a sequence point t with l.t <= t.t <= r.t,
 * returns an interpolated signal point at time t.t.
 */
__device__ sigpt_t
interpolate(const sigpt_t *l,
            const sigpt_t *r,
            const seqpt_t *t)
{
    const sigpt_t l_reg = *l;
    const sigpt_t r_reg = *r;

    const float dt = r_reg.t - l_reg.t;
    const float dy = r_reg.y - l_reg.y;
    const float dy_normed = dy / dt; /* TODO: Assumes dt != 0.f. */

    sigpt_t sigpt = { t->t, l_reg.y + dy_normed * (t->t - l_reg.t), dy_normed };
    return sigpt;
}

__global__ void
sigpt_extrapolate(const sigpt_t *lhs,
                  const sigpt_t *rhs,
                  const seqpt_t *ts,
                  sigpt_t *lhs_extrapolated,
                  sigpt_t *rhs_extrapolated,
                  int n_lhs,
                  int n_rhs,
                  int n_ts)
{
    /* Use the information provided by lhs, rhs, and ts
     * to extrapolated a signal point sequence for both lhs and rhs for each
     * time point in ts. */

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n_ts; i += blockDim.x * gridDim.x) {
        const seqpt_t seqpt = ts[i];

        const int is_lhs = (seqpt.flags & FLAG_LHS) != 0;
        const int is_rhs = !is_lhs;

        const int assoc_lhs = is_lhs * seqpt.i + is_rhs * seqpt.assoc_i;
        const int assoc_rhs = is_rhs * seqpt.i + is_lhs * seqpt.assoc_i;

        /* TODO: Optimize. */

        if (assoc_lhs >= n_lhs - 1) {
            lhs_extrapolated[i] = (sigpt_t){ seqpt.t, lhs[i].y, lhs[i].dy };
        } else {
            lhs_extrapolated[i] = interpolate(lhs + assoc_lhs,
                                              lhs + assoc_lhs + 1,
                                              &seqpt);
        }

        if (assoc_rhs >= n_rhs - 1) {
            rhs_extrapolated[i] = (sigpt_t){ seqpt.t, rhs[i].y, rhs[i].dy };
        } else {
            rhs_extrapolated[i] = interpolate(rhs + assoc_rhs,
                                              rhs + assoc_rhs + 1,
                                              &seqpt);
        }
    }
}

__global__ void
extract_i(const seqpt_t *in, int *out, int n, int flag)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        seqpt_t s = in[i];
        const int has_flag = (s.flags & flag) != 0;
        out[i] = has_flag * s.i;
    }
}

__global__ void
merge_i(const int *lhs, const int *rhs, seqpt_t *out, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        const int is_lhs = (out[i].flags & FLAG_LHS) != 0;
        const int is_rhs = !is_lhs;

        out[i].assoc_i = is_lhs * rhs[i] + is_rhs * lhs[i];
    }
}

__global__ void
sigpt_to_seqpt(const sigpt_t *in, seqpt_t *out, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        out[i].t = in[i].t;
        out[i].i = i;
    }
}

__global__ void
insert_proto_intersections(const seqpt_t *in, seqpt_t *out, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        seqpt_t seqpt = in[i];
        out[i * 2] = seqpt;

        seqpt.flags |= FLAG_ISC;
        out[i * 2 + 1] = seqpt;
    }
}

__global__ void
calc_intersections(const sigpt_t *lhs,
                   const sigpt_t *rhs,
                   seqpt_t *ts,
                   int n_lhs,
                   int n_rhs,
                   int n_ts)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    /* At this point, we are only interested in intersection elements in ts.
     * These are located at every index 2 * i + 1, i <- N.
     *
     * Assuming ts.flags ~ FLAG_LHS: ts[i].i is the index of the current
     * point in lhs (the next point is obviously lhs[ts[i].i + 1]).
     * The closest point to the left in in rhs is located at index
     * rhs_max[i], and the closest to the right is rhs_max[i] + 1.
     *
     * This is enough information to determine the time of the signal 
     * intersection.
     */

    for (int i = tid; 2 * i + 1 < n_ts; i += blockDim.x * gridDim.x) {
        const int ii = 2 * i + 1;
        seqpt_t s = ts[ii];

        const int is_lhs = (s.flags & FLAG_LHS) != 0;
        const int is_rhs = !is_lhs;

        /* TODO: Optimize. */
        const sigpt_t *this_sig = is_lhs ? lhs : rhs;
        const sigpt_t *other_sig = is_lhs ? rhs : lhs;

        /* We now have four points corresponding to the end points of the
         * two line segments. (x1, y1) and (x2, y2) for one line segment,
         * (x3, y3) and (x4, y4) for the other line segment.
         * We are interested in the x coordinate of the intersection:
         * x = ((x1y2 - y1x2)(x3 - x4) - (x1 - x2)(x3y4 - y3x4)) /
         *     ((x1 - x2)(y3 - y4) - (y1 - y2)(x3 - x4)).
         * If the denominator is 0, the lines are parallel. We only
         * care about intersections in a specific interval - if 
         * there is none, we mark the element with FLAG_DEL.
         */

        if ((is_lhs && (s.i >= n_lhs - 2 || s.assoc_i >= n_rhs - 2)) ||
            (is_rhs && (s.i >= n_rhs - 2 || s.assoc_i >= n_lhs - 2))) {
            continue; /* TODO: Optimize */
        }

        const sigpt_t p1 = this_sig[s.i];
        const sigpt_t p2 = this_sig[s.i + 1];
        const sigpt_t p3 = other_sig[s.assoc_i];
        const sigpt_t p4 = other_sig[s.assoc_i + 1];

        const float denom = (p1.t - p2.t) * (p3.y - p4.y) -
                            (p1.y - p2.y) * (p3.t - p4.t);
        const float numer = (p1.t * p2.y - p1.y * p2.t) * (p3.t - p4.t) -
                            (p1.t - p2.t) * (p3.t * p4.y - p3.y * p4.t);

        /* Lines parallel? */
        if (denom == 0.f) {
            continue; /* TODO: Optimize */
        }

        const float t = numer / denom;

        /* Intersection outside of line segment range? */
        if (t <= p1.t || t >= p2.t || t <= p3.t || t >= p4.t) {
            continue; /* TODO: Optimize */
        }

        ts[ii].t = t;
    }
}

struct seqpt_same_time : public thrust::binary_function<seqpt_t, seqpt_t, bool>
{
    __device__ bool
    operator()(const seqpt_t &lhs, const seqpt_t &rhs) const
    {
        return abs(lhs.t - rhs.t) < FLOAT_DELTA;
    }
};


/**
 * sizeof(out) = 4 * max(sizeof(lhs), sizeof(rhs)).
 */
void
stl_and(const thrust::device_vector<sigpt_t> &lhs,
        const thrust::device_vector<sigpt_t> &rhs,
        thrust::device_vector<sigpt_t> out)
{
    /* A rough outline of the function:
     *
     * Construct a sorted time sequence ts
     * which contains all t <- lhs, all t <- rhs, and all intersection points
     * between lhs and rhs. The sequence contains only unique points.
     * 
     * Using interpolation, construct lhs' and rhs' such that they contain all
     * t <- ts.
     *
     * Finally, do a simple min() over these arrays.
     */

    /* First, extract the time sequences, merge them, and remove duplicates. */

    const sigpt_t *ptr_lhs = thrust::raw_pointer_cast(lhs.data());
    seqpt_t seqinit = { 0.f, 0, 0, FLAG_LHS };
    thrust::device_vector<seqpt_t> lhs_ts(lhs.size(), seqinit);
    seqpt_t *ptr_lhs_ts = thrust::raw_pointer_cast(lhs_ts.data());
    sigpt_to_seqpt<<<NBLOCKS, NTHREADS>>>(ptr_lhs, ptr_lhs_ts, lhs.size());

    const sigpt_t *ptr_rhs = thrust::raw_pointer_cast(rhs.data());
    seqinit.flags = FLAG_RHS;
    thrust::device_vector<seqpt_t> rhs_ts(rhs.size(), seqinit);
    seqpt_t *ptr_rhs_ts = thrust::raw_pointer_cast(rhs_ts.data());
    sigpt_to_seqpt<<<NBLOCKS, NTHREADS>>>(ptr_rhs, ptr_rhs_ts, rhs.size());

    thrust::device_vector<seqpt_t> ts(lhs_ts.size() + rhs_ts.size(), seqinit);
    thrust::merge(lhs_ts.begin(), lhs_ts.end(), rhs_ts.begin(), rhs_ts.end(),
                  ts.begin(), seqpt_less());

    thrust::device_vector<seqpt_t>::iterator ts_end =
        thrust::unique(ts.begin(), ts.end(), seqpt_same_time());
    int ts_size = ts_end - ts.begin();

    /* Add a proto-intersection after each point in the resulting sequence. */

    seqinit.flags = 0;
    const seqpt_t *ptr_ts = thrust::raw_pointer_cast(ts.data());
    thrust::device_vector<seqpt_t> tsi(ts_size * 2, seqinit);
    seqpt_t *ptr_tsi = thrust::raw_pointer_cast(tsi.data());
    insert_proto_intersections<<<NBLOCKS, NTHREADS>>>(ptr_ts, ptr_tsi, ts_size);

    /* Now, for every proto-intersection of side s <- { lhs, rhs }, we need to
     * find the index of the element to its immediate left of the opposing side.
     * We do this by first extracting the indices of each side to an array,
     * running a max() scan over it, and finally merging these arrays back into
     * seqpt_t.assoc_i.
     */ 

    thrust::device_vector<int> lhs_i_max(tsi.size(), 0);
    int *ptr_lhs_i_max = thrust::raw_pointer_cast(lhs_i_max.data());
    extract_i<<<NBLOCKS, NTHREADS>>>(ptr_tsi, ptr_lhs_i_max, tsi.size(), FLAG_LHS);

    thrust::inclusive_scan(lhs_i_max.begin(), lhs_i_max.end(), lhs_i_max.begin(),
                           thrust::maximum<int>());

    thrust::device_vector<int> rhs_i_max(tsi.size(), 0);
    int *ptr_rhs_i_max = thrust::raw_pointer_cast(rhs_i_max.data());
    extract_i<<<NBLOCKS, NTHREADS>>>(ptr_tsi, ptr_rhs_i_max, tsi.size(), FLAG_RHS);

    thrust::inclusive_scan(rhs_i_max.begin(), rhs_i_max.end(), rhs_i_max.begin(),
                           thrust::maximum<int>());

    merge_i<<<NBLOCKS, NTHREADS>>>(ptr_lhs_i_max, ptr_rhs_i_max, ptr_tsi, tsi.size());

    /* Next, we go through and fill in ISC elements; if there's an intersection
     * we set the time accordingly, and if there isn't, we mark it as DEL.
     * An intersection must always be between the latest (lhs, rhs) and the next such
     * pair.
     */

    calc_intersections<<<NBLOCKS, NTHREADS>>>(ptr_lhs, ptr_rhs, ptr_tsi,
                                              lhs.size(), rhs.size(), tsi.size());

    /* Finally we again remove all duplicate elements (= all proto-intersections
     * which did not turn out to actually be real intersections).
     */

    thrust::device_vector<seqpt_t>::iterator tsi_end =
        thrust::unique(tsi.begin(), tsi.end(), seqpt_same_time());
    int tsi_size = ts_end - ts.begin();

    /* We now have the complete time sequence stored in ts, including
     * all points in lhs, rhs, and intersections of the two (what a bitch).
     * Extrapolate the sigpt_t sequence of both signals for each point <- ts.
     */

    sigpt_t sigpt_init = { 0.f, 0.f, 0.f };
    thrust::device_vector<sigpt_t> lhs_extrapolated(tsi_size, sigpt_init);
    thrust::device_vector<sigpt_t> rhs_extrapolated(tsi_size, sigpt_init);
    sigpt_extrapolate<<<NBLOCKS, NTHREADS>>>(ptr_lhs, ptr_rhs, ptr_tsi,
            thrust::raw_pointer_cast(lhs_extrapolated.data()),
            thrust::raw_pointer_cast(rhs_extrapolated.data()),
            lhs.size(), rhs.size(), tsi_size);

    /* And *finally* run the actual and operator. */

    thrust::transform(lhs_extrapolated.begin(), lhs_extrapolated.end(),
            rhs_extrapolated.begin(),
            out.begin(),
            sigpt_min());

    /* TODO: Instead of allocating all of these device vectors between 
     * kernel calls, try to be a bit smarter about it. For example,
     * we could queue the allocations on a separate stream. */

    printf("lhs (%d):\n", lhs.size());
    for (int i = 0; i < 10; i++) {
        sigpt_t sigpt = lhs[i];
    	printf("%i: {%f, %f, %f}\n", i, sigpt.t, sigpt.y, sigpt.dy);
    }

    printf("\nrhs (%d):\n", rhs.size());
    for (int i = 0; i < 10; i++) {
        sigpt_t sigpt = rhs[i];
    	printf("%i: {%f, %f, %f}\n", i, sigpt.t, sigpt.y, sigpt.dy);
    }

    printf("\ntsi (%d):\n", tsi_size);
    for (int i = 0; i < 10; i++) {
    	seqpt_t s = tsi[i];
    	printf("{ %f, %d, %d, %x }\n", s.t, s.i, s.assoc_i, s.flags);
    }

    printf("\nlhs_extrapolated (%d):\n", lhs_extrapolated.size());
    for (int i = 0; i < 10; i++) {
        sigpt_t sigpt = lhs_extrapolated[i];
    	printf("%i: {%f, %f, %f}\n", i, sigpt.t, sigpt.y, sigpt.dy);
    }

    printf("\nrhs_extrapolated (%d):\n", rhs_extrapolated.size());
    for (int i = 0; i < 10; i++) {
        sigpt_t sigpt = rhs_extrapolated[i];
    	printf("%i: {%f, %f, %f}\n", i, sigpt.t, sigpt.y, sigpt.dy);
    }
}

struct sigpt_max : public thrust::binary_function<sigpt_t, sigpt_t, sigpt_t>
{
    __device__ sigpt_t
    operator()(const sigpt_t &lhs, const sigpt_t &rhs) const
    {
        /*sigpt_t ret;
        if (lhs.y > rhs.y) {
            ret.y = lhs.y;
        } else {
            ret.y = rhs.y;
        }

        // Keep the time.
        ret.t = rhs.t;

        return ret;*/

        return (sigpt_t) {rhs.t, CUDA_MAX(lhs.y, rhs.y), 0};
    }
};

__global__ void
eventually_intersect(const sigpt_t *ys, sigpt_t *zs, sigpt_t *zs_intersect, char *cs, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        cs[i * 2] = 1;
	zs_intersect[i * 2] = zs[i];
        // FIXME: Branches are bad.
        if (i < n - 1 && zs[i].y > zs[i + 1].y) {
            cs[i * 2 + 1] = 1;
            zs_intersect[i * 2 + 1].t = zs[i + 1].t +
		    (zs[i + 1].t - zs[i].t) *
		    (zs[i + 1].y - ys[i + 1].y) / (ys[i + 1].y - ys[i].y);
            zs_intersect[i * 2 + 1].y = zs[i + 1].y;
        }
    }
}

__global__ void
eventually_compact(const sigpt_t *zs, sigpt_t *zs_final, const char *cs, const size_t *fs, int n)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // FIXME: Branches are bad.
        if (cs[i] == 1) {
            zs_final[fs[i]] = zs[i];
        }
    }
}

/**
 * sizeof(out) = 2 * sizeof(in).
 */
void
stl_eventually(const thrust::device_vector<sigpt_t> &in,
        thrust::device_vector<sigpt_t> &out)
{
    thrust::inclusive_scan(in.crbegin(), in.crend(), out.rbegin() + in.size(), sigpt_max()); 

    const sigpt_t *ys = thrust::raw_pointer_cast(in.data());

    thrust::device_vector<sigpt_t> out_intersect(in.size() * 2);
    sigpt_t *zs = thrust::raw_pointer_cast(out_intersect.data());

    sigpt_t *zs_final = thrust::raw_pointer_cast(out.data());

    thrust::device_vector<char> used(in.size() * 2, 0);
    char *cs = thrust::raw_pointer_cast(used.data());
    eventually_intersect<<<NBLOCKS, NTHREADS>>>(ys, zs_final, zs, cs, in.size());

    thrust::device_vector<size_t> positions(in.size() * 2, 0);
    thrust::exclusive_scan(used.cbegin(), used.cend(), positions.begin(), 0, thrust::plus<size_t>()); 

    size_t *fs = thrust::raw_pointer_cast(positions.data());
    eventually_compact<<<NBLOCKS, NTHREADS>>>(zs, zs_final, cs, fs, in.size() * 2);

    out.resize(positions.back());
}

int
main(int argc, char **argv)
{
    sigpt_t *a = sigpt_random(42, NITEMS);
    sigpt_t *b = sigpt_random(43, NITEMS);
    sigpt_t *c = (sigpt_t *)calloc(4 * NITEMS,sizeof(sigpt_t));

    thrust::device_vector<sigpt_t> lhs(a, a + NITEMS);
    thrust::device_vector<sigpt_t> rhs(b, b + NITEMS);
    thrust::device_vector<sigpt_t> out(c, c + 4 * NITEMS);

    stl_and(lhs, rhs, out);

    free(a);
    free(b);
    free(c);

    /* And a Thrust scan operation, let's see how we can integrate that with the rest of
     * the code...
     */

    /*

    thrust::plus<float> binary_op;

    thrust::host_vector<float> hostVector(NITEMS);
    thrust::generate(hostVector.begin(), hostVector.end(), rand);

    thrust::device_vector<float> deviceVector = hostVector;
    thrust::exclusive_scan(deviceVector.begin(), deviceVector.end(), deviceVector.begin(), 0.f, binary_op);

    thrust::copy(deviceVector.begin(), deviceVector.end(), hostVector.begin());

    for (thrust::device_vector<float>::iterator iter = deviceVector.begin();
         iter != deviceVector.begin() + 10;
         iter++) {
    	float val = *iter;
    	printf("%f ", val);
    }
    printf("\n");

    */

    return 0;
}
