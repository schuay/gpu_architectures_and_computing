#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>

extern "C" {
#include "util.h"
}

#define NBLOCKS (256)
#define NTHREADS (256)

#define NITEMS (256 * 257)

#define FLOAT_DELTA (0.000000001f)

#define checkCudaError(val) do { _checkCudaError((val), #val, __FILE__, __LINE__); } while (0)

bool
_checkCudaError(hipError_t result, const char *func, const char *file, int line)
{
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        return true;
    } else {
        return false;
    }
}

__global__ void
simpleAnd(const float *lhs, const float *rhs, float *result, int nitems)
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = tid; i < nitems; i += blockDim.x * gridDim.x) {
		result[i] = ((lhs[i] + rhs[i]) >= 2.f - FLOAT_DELTA) ? 1.f : 0.f;
	}
}

__global__ void
simpleOr(const float *lhs, const float *rhs, float *result, int nitems)
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = tid; i < nitems; i += blockDim.x * gridDim.x) {
		result[i] = ((lhs[i] + rhs[i]) >= 1.f - FLOAT_DELTA) ? 1.f : 0.f;
	}
}

int
main(int argc, char **argv)
{
    float *a = random_array(42, NITEMS);
    float *b = random_array(43, NITEMS);
    float *c = (float *)calloc(NITEMS, sizeof(float));
    float *devA, *devB, *devC;

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream));

    checkCudaError(hipMalloc((void **)&devA, NITEMS * sizeof(float)));
    checkCudaError(hipMalloc((void **)&devB, NITEMS * sizeof(float)));
    checkCudaError(hipMalloc((void **)&devC, NITEMS * sizeof(float)));

    checkCudaError(hipMemcpy(devA, a, NITEMS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(devB, b, NITEMS * sizeof(float), hipMemcpyHostToDevice));

    printf("Launching kernel...\n");

    /**
     * These calls are asynchronous and just queue the pending operations up on the stream.
     * We can queue our entire operator tree here and run it without interruption from the host.
     */
    simpleOr<<<NBLOCKS, NTHREADS, 0, stream>>>(devA, devB, devC, NITEMS);
    simpleAnd<<<NBLOCKS, NTHREADS, 0, stream>>>(devC, devB, devC, NITEMS);

    checkCudaError(hipMemcpy(a, devA, NITEMS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(b, devB, NITEMS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(c, devC, NITEMS * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < 20; i++) {
    	printf("%f & %f = %f\n", a[i], b[i], c[i]);
    }

    checkCudaError(hipFree(devA));
    checkCudaError(hipFree(devB));
    checkCudaError(hipFree(devC));

    free(a);
    free(b);
    free(c);

    return 0;
}
